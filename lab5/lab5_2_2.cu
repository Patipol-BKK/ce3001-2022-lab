#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void cudaVectorAddition(int *vecC, int *vecA, int *vecB, int n) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	vecC[idx] = vecA[idx] + vecB[idx];
}

int main() {
	int n = 4;
	int vecA[n] = {1, 2, 4, 2};
	int vecB[n] = {7, 1, 3, 5};
	int vecC[n];

	int *dVecA, *dVecB, *dVecC;
	hipMalloc((void**)&dVecA, sizeof(int)*n);
	hipMalloc((void**)&dVecB, sizeof(int)*n);
	hipMalloc((void**)&dVecC, sizeof(int)*n);

	hipMemcpy(dVecA, vecA, sizeof(int)*n, hipMemcpyHostToDevice);
	hipMemcpy(dVecB, vecB, sizeof(int)*n, hipMemcpyHostToDevice);

	cudaVectorAddition<<<2, 4>>>(dVecC, dVecA, dVecB, n);
	hipDeviceSynchronize();
	hipMemcpy(vecC, dVecC, sizeof(int)*n, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	hipFree(dVecA);
	hipFree(dVecB);
	hipFree(dVecC);

	int i;
	printf("A   ");
	for(i = 0; i < n; i++) {
		printf("%2d ", vecA[i]);
	}
	printf("\n");

	printf("B   ");
	for(i = 0; i < n; i++) {
		printf("%2d ", vecB[i]);
	}
	printf("\n");

	printf("C   ");
	for(i = 0; i < n; i++) {
		printf("%2d ", vecC[i]);
	}
	printf("\n");

	return 0;
}