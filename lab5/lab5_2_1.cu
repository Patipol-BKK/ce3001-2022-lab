
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_GPU(int id) {
	int i = threadIdx.x;
	printf("Hello from GPU%d[%d]!\n", id, i);
}


int main(void) {
	printf("Hello from CPU!\n");
	hello_GPU<<<1,4>>>(1);
	hello_GPU<<<1,6>>>(2);
	hipDeviceSynchronize();
	return 0;
}