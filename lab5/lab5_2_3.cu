#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void cudaVectorDotProduct(int *vecC, int *vecA, int *vecB, int n) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int i;
	vecC[idx] = vecA[idx] * vecB[idx];
	__syncthreads();
	if(!idx) {
		for(i = 1; i < n; i++) {
			vecC[idx] += vecC[i];
		}
	}
}

int main() {
	int n = 4;
	int vecA[n] = {22, 13, 16, 5};
	int vecB[n] = {5, 22, 17, 37};
	int vecC[n];

	int *dVecA, *dVecB, *dVecC;
	hipMalloc((void**)&dVecA, sizeof(int)*n);
	hipMalloc((void**)&dVecB, sizeof(int)*n);
	hipMalloc((void**)&dVecC, sizeof(int)*n);

	hipMemcpy(dVecA, vecA, sizeof(int)*n, hipMemcpyHostToDevice);
	hipMemcpy(dVecB, vecB, sizeof(int)*n, hipMemcpyHostToDevice);

	cudaVectorDotProduct<<<2, 4>>>(dVecC, dVecA, dVecB, n);
	hipDeviceSynchronize();
	hipMemcpy(vecC, dVecC, sizeof(int)*n, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	hipFree(dVecA);
	hipFree(dVecB);
	hipFree(dVecC);

	int i;
	printf("A   ");
	for(i = 0; i < n; i++) {
		printf("%2d ", vecA[i]);
	}
	printf("\n");

	printf("B   ");
	for(i = 0; i < n; i++) {
		printf("%2d ", vecB[i]);
	}
	printf("\n");

	printf("Answer = %d\n", vecC[0]);
	
	return 0;
}